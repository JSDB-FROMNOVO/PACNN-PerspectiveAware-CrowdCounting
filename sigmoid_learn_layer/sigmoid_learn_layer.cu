#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/sigmoid_learn_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out, Dtype rSlope, Dtype bias) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 0.5 * tanh(0.5 * (rSlope * (in[index] - bias)) ) + 0.5;
  }
}

template <typename Dtype>
void SigmoidLearnLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data_1 = top[0]->mutable_gpu_data();
  Dtype* top_data_2 = top[1]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const Dtype* rSlope = this->blobs_[0]->cpu_data();
  const Dtype* bias = this->blobs_[1]->cpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data_1, rSlope[0], bias[0]);
  CUDA_POST_KERNEL_CHECK;
  
  // test
  //LOG(INFO) << "0000000000000000000" << "\n";
  const Dtype* top_data_cpu = top[0]->cpu_data();
  const Dtype* bottom_data_cpu = bottom[0]->cpu_data();
  for (int i = 0; i < 10; i++){
  //LOG(INFO) << "forward top data " << top_data_cpu[i] << "\n";
  //LOG(INFO) << "forward bottom data " << bottom_data_cpu[i] << "\n";
  }
  //LOG(INFO) << "0000000000000000000" << "\n";

  Blob<Dtype>* ones = new Blob<Dtype>();
  ones->ReshapeLike(*top[0]);
  caffe_gpu_set(int(top[0]->count()), Dtype(1), ones->mutable_gpu_data());
  caffe_gpu_sub(int(top[0]->count()), ones->gpu_data(), top[0]->gpu_data(), top[1]->mutable_gpu_data());
  
  //LOG(INFO) << "rSlope: " << rSlope[0] << ", bias: " << bias[0] << ", top0: " << top[0]->cpu_data()[0] << ", top1: " << top[1]->cpu_data()[0] << "\n";
  ////LOG(INFO) << "----------------" << "\n";
  ////LOG(INFO) << "rSlope		" << rSlope[0] << "\n";
  ////LOG(INFO) << "bias		" << bias[0] << "\n";
  ////LOG(INFO) << "bottom		" << bottom[0]->cpu_data()[0] << "\n"; 
  ////LOG(INFO) << "top0		" << top[0]->cpu_data()[0] << "\n"; 
  ////LOG(INFO) << "top1		" << top[1]->cpu_data()[0] << "\n"; 
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidBackward(const int n, const Dtype* out_diff_1, const Dtype* out_diff_2,
    const Dtype* out_data, Dtype* in_diff, Dtype rSlope) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    in_diff[index] = rSlope * (out_diff_1[index] - out_diff_2[index]) * sigmoid_x * (1 - sigmoid_x);
  }
}

template <typename Dtype>
__global__ void SigmoidParamBackward(const int n, const Dtype* out_diff_1, const Dtype* out_diff_2,
    const Dtype* out_data, const Dtype* in_data,
    Dtype* bias_diff, Dtype* rSlope_diff, Dtype rSlope, Dtype bias) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    rSlope_diff[index] = (in_data[index] - bias) * (out_diff_1[index] - out_diff_2[index]) * sigmoid_x * (1 - sigmoid_x);
    bias_diff[index] = (-1.) * rSlope * (out_diff_1[index] - out_diff_2[index]) * sigmoid_x * (1 - sigmoid_x);
    //bias_diff[0] += 1;
  }
}

template <typename Dtype>
void SigmoidLearnLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* rSlope = this->blobs_[0]->cpu_data();
  const Dtype* bias = this->blobs_[1]->cpu_data();
  if (this->param_propagate_down_[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff_1 = top[0]->gpu_diff();
    const Dtype* top_diff_2 = top[1]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    
    // test
    //LOG(INFO) << "!!!" << "\n";
    const Dtype* bias_diff_1 = this->blobs_[1]->cpu_diff();
    //LOG(INFO) << "bias shape " << this->blobs_[1]->shape_string() << "\n";
    //LOG(INFO) << "bias_diff " << bias_diff_1[0] << "\n";
    //LOG(INFO) << "!!!" << "\n";

    Dtype* rSlope_diff = this->blobs_[0]->mutable_cpu_diff();
    Dtype* bias_diff = this->blobs_[1]->mutable_cpu_diff();
    // change
    SigmoidParamBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff_1, top_diff_2, top_data, bottom_data, bias_diff_mat_.mutable_gpu_data(), rSlope_diff_mat_.mutable_gpu_data(), rSlope[0], bias[0]);  
    Dtype sum = 0;
    const Dtype* bias_diff_gpu = bias_diff_mat_.gpu_data();
    const Dtype* rSlope_diff_gpu = rSlope_diff_mat_.gpu_data();
    Blob<Dtype>* ones = new Blob<Dtype>();
    ones->ReshapeLike(bias_diff_mat_);
    caffe_gpu_set(int(ones->count()), Dtype(1), ones->mutable_gpu_data());
    caffe_gpu_dot(int(ones->count()), ones->gpu_data(), bias_diff_mat_.gpu_data(), &sum); 
    bias_diff[0] = sum;
    caffe_gpu_dot(int(ones->count()), ones->gpu_data(), rSlope_diff_mat_.gpu_data(), &sum); 
    rSlope_diff[0] = sum;
    
        

    // test
    //LOG(INFO) << "!!!" << "\n";
    const Dtype* bias_diff_2 = this->blobs_[1]->cpu_diff();
    //LOG(INFO) << "after backward bias_diff " << bias_diff_2[0] << "\n";
    //LOG(INFO) << "!!!" << "\n";
    // test 
    const Dtype* top_diff_1_1 = top[0]->cpu_diff();
    const Dtype* top_diff_2_2 = top[1]->cpu_diff();
    //LOG(INFO) << "================" << "\n";
    for (int i = 0; i <10 ; i++){
    //LOG(INFO) << "top_diff_1 " << top_diff_1_1[i] << "\n"; 
    //LOG(INFO) << "top_diff_2 " << top_diff_2_2[i] << "\n"; 
    }
    //LOG(INFO) << "================" << "\n";
    const Dtype* bias_diff_mat_cpu = bias_diff_mat_.cpu_data();
    for (int i = 0; i<10 ; i++){
    //LOG(INFO) << "bias_diff_mat '" << bias_diff_mat_cpu[i] << "\n";
    }
    //LOG(INFO) << "================" << "\n";
    const Dtype* top_data_cpu = top[0]->cpu_data();
    for (int i = 0; i<10; i++){
    //LOG(INFO) << "top_data " << top_data_cpu[i] << "\n";
    }
    //LOG(INFO) << "================" << "\n";
  }
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff_1 = top[0]->gpu_diff();
    const Dtype* top_diff_2 = top[1]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff_1, top_diff_2, top_data, bottom_diff, rSlope[0]);
    //const Dtype* bias_diff = bias_diff_mat_.cpu_data();
    //const Dtype* rSlope_diff = rSlope_diff_mat_.cpu_data();
    //for (int i = 0; i < int(bias_diff_mat_.count()); i++) {
    //  bias_ -= (bias_diff[i] * bias_lr_mult_);
    //  rSlope_ -= (rSlope_diff[i] * rSlope_lr_mult_);
    //}
    CUDA_POST_KERNEL_CHECK;
  }
    const Dtype* r_diff = this->blobs_[0]->cpu_diff();
    const Dtype* b_diff = this->blobs_[1]->cpu_diff();
    ////LOG(INFO) << "bias diff	"<< b_diff[0] << "\n";
    ////LOG(INFO) << "rSlope diff 	"<< r_diff[0] << "\n";
    ////LOG(INFO) << "bottom diff	"<< bottom[0]->cpu_diff()[0] << "\n";
    ////LOG(INFO) << "top0 diff	"<< top[0]->cpu_diff()[0]  << "\n";
    ////LOG(INFO) << "top1 diff	"<< top[1]->cpu_diff()[0]  << "\n";
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidLearnLayer);


}  // namespace caffe
